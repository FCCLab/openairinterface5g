/*
 * Copyright 1993-2024 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */

// #include "../common/book.h"
#include <stdio.h>
#include <hip/hip_runtime.h>


#define SIZE (64 * 1024 * 1024)

#define HANDLE_ERROR(x)                                                                 \
    do                                                                                  \
    {                                                                                   \
        if((x) != hipSuccess) { printf("Error %s line%d\n", __FUNCTION__, __LINE__); } \
    } while(0)
#define HANDLE_NULL(x)

float cuda_malloc_test(int size, bool up)
{
    hipEvent_t start, stop;
    int *       a = NULL, *dev_a = NULL;
    float       elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    a = (int*)malloc(size * sizeof(*a));
    if (a == NULL)
    {
        return 0.0;
    }
    memset(a, 0, size * sizeof(*a));
    HANDLE_NULL(a);
    HANDLE_ERROR(hipMalloc((void**)&dev_a, size * sizeof(*dev_a)));

    HANDLE_ERROR(hipEventRecord(start, 0));
    for(int i = 0; i < 100; i++)
    {
        if(up)
            HANDLE_ERROR(
                hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice));
        else
            HANDLE_ERROR(
                hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost));
    }
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    free(a);
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    return elapsedTime;
}

float cuda_host_register_test(int size, bool up)
{
    hipEvent_t start, stop;
    int *       a, *dev_a;
    float       elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    a = (int*)malloc(size * sizeof(*a));
    if (a == NULL)
    {
        return 0.0;
    }
    memset(a, 0, size * sizeof(*a));
    HANDLE_NULL(a);

    int flag = hipHostRegisterPortable | hipHostRegisterMapped;
    HANDLE_ERROR(hipHostRegister(a, size * sizeof(*a), flag));

    HANDLE_ERROR(hipMalloc((void**)&dev_a, size * sizeof(*dev_a)));

    HANDLE_ERROR(hipEventRecord(start, 0));
    for(int i = 0; i < 100; i++)
    {
        if(up)
            HANDLE_ERROR(
                hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice));
        else
            HANDLE_ERROR(
                hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost));
    }
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    HANDLE_ERROR(hipHostUnregister(a));
    free(a);
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    return elapsedTime;
}

float cuda_host_alloc_test(int size, bool up)
{
    hipEvent_t start, stop;
    int *       a, *dev_a;
    float       elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    HANDLE_ERROR(
        hipHostAlloc((void**)&a, size * sizeof(*a), hipHostMallocDefault));
    HANDLE_ERROR(hipMalloc((void**)&dev_a, size * sizeof(*dev_a)));

    HANDLE_ERROR(hipEventRecord(start, 0));
    for(int i = 0; i < 100; i++)
    {
        if(up)
            HANDLE_ERROR(
                hipMemcpy(dev_a, a, size * sizeof(*a), hipMemcpyHostToDevice));
        else
            HANDLE_ERROR(
                hipMemcpy(a, dev_a, size * sizeof(*a), hipMemcpyDeviceToHost));
    }
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    HANDLE_ERROR(hipHostFree(a));
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    return elapsedTime;
}

int main(void)
{
    float elapsedTime;
    float MB = (float)100 * SIZE * sizeof(int) / 1024 / 1024;

    // try it with cudaMalloc
    elapsedTime = cuda_malloc_test(SIZE, true);
    printf("Time using cudaMalloc:  %3.1f ms", elapsedTime);
    printf("\tMB/s during copy up:  %3.1f\n", MB / (elapsedTime / 1000));

    elapsedTime = cuda_malloc_test(SIZE, false);
    printf("Time using cudaMalloc:  %3.1f ms", elapsedTime);
    printf("\tMB/s during copy down:  %3.1f\n", MB / (elapsedTime / 1000));

    // now try it with cudaHostAlloc
    elapsedTime = cuda_host_alloc_test(SIZE, true);
    printf("Time using cudaHostAlloc:  %3.1f ms", elapsedTime);
    printf("\tMB/s during copy up:  %3.1f\n", MB / (elapsedTime / 1000));

    elapsedTime = cuda_host_alloc_test(SIZE, false);
    printf("Time using cudaHostAlloc:  %3.1f ms", elapsedTime);
    printf("\tMB/s during copy down:  %3.1f\n", MB / (elapsedTime / 1000));

    // try it with cudaMalloc
    elapsedTime = cuda_host_register_test(SIZE, true);
    printf("Time using cudaHostRegister:  %3.1f ms", elapsedTime);
    printf("\tMB/s during copy up:  %3.1f\n", MB / (elapsedTime / 1000));

    elapsedTime = cuda_host_register_test(SIZE, false);
    printf("Time using cudaHostRegister:  %3.1f ms", elapsedTime);
    printf("\tMB/s during copy down:  %3.1f\n", MB / (elapsedTime / 1000));
}
