#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime_api.h>

#include "test_cuda.h"
#include "nv_ipc_utils.h"

#define TAG "NVIPC.TESTCUDA"

#define N_BLOCK 64
#define N_THREAD 32;

static const char SUB_VAL = (char) ('A' - 'a');

inline hipError_t __checkLastCudaError(const char* file, int line)
{
    hipError_t lastErr = hipGetLastError();
    if(lastErr != hipSuccess)
    {
        NVLOGE_FMT(TAG, AERIAL_NVIPC_API_EVENT, "Error at {} line {}: {}", file, line, hipGetErrorString(lastErr));
    }
    return lastErr;
}

#define checkLastCudaError() __checkLastCudaError(__FILE__, __LINE__)

#define HANDLE_ERROR(x)                                                                 \
    do                                                                                  \
    {                                                                                   \
        if((x) != hipSuccess) { printf("Error %s line%d\n", __FUNCTION__, __LINE__); } \
    } while(0)
#define HANDLE_NULL(x)

static __global__ void gpu_to_lower_case(char* str, int length)
{
    int index  = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < length; i += stride)
    {
        if(str[i] >= 'A' && str[i] <= 'Z')
        {
            str[i] -= SUB_VAL;
        }
    }
}

void test_cuda_to_lower_case(int deviceId, char* str, int length, int gpu)
{
    if(deviceId < 0)
    {
        NVLOGD_FMT(TAG, "{}: deviceId={}, fall back to CPU IPC test", __func__, deviceId);
        gpu = 0;
    }

    NVLOGI_FMT(TAG, "{}: gpu={}", __func__, gpu);
    if(gpu)
    {
        int nblock  = N_BLOCK;
        int nthread = N_THREAD;
        HANDLE_ERROR(hipSetDevice(deviceId));
        gpu_to_lower_case<<<nblock, nthread>>>(str, length);
        checkLastCudaError();
    }
    else
    {
        cpu_to_lower_case(str, length);
    }
}

int get_cuda_device_id(void)
{
    int num;
    hipError_t err = hipGetDeviceCount (&num);
    NVLOGC_FMT(TAG, "{}: err={} num={}", __func__, err, num);

    if (err == hipSuccess && num > 0)
    {
        return 0;
    }
    else
    {
        return -1;
    }
}

void cuda_to_lower_case(char* str, int length, int deviceId)
{
    if(deviceId < 0)
    {
        NVLOGC_FMT(TAG, "{}: invalid CUDA deviceId: {}", __func__, deviceId);
        return;
    }

    int nblock  = N_BLOCK;
    int nthread = N_THREAD;
    HANDLE_ERROR(hipSetDevice(deviceId));
    gpu_to_lower_case<<<nblock, nthread>>>(str, length);
    checkLastCudaError();
}
