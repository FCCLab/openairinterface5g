/*
 * Copyright (c) 2020-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

#include <stdio.h>
#include <string.h>
#include <sys/mman.h>
#include "hip/hip_runtime.h"
#include "gdrapi.h"
#include "nv_ipc_gpudata_utils.h"
#include "nv_ipc_utils.h"

static char TAG[]      = "NVIPC.GPUDATAUTILS";

#if 0
#define CUDA_CHECK( fn ) do { \
  hipError_t status = (fn); \
  if ( hipSuccess != status ) { \
    const char* errstr; \
    hipDrvGetErrorString(status, &errstr); \
    NVLOGE_NO(TAG, AERIAL_CUDA_API_EVENT, "CUDA Driver Failure (line %d of file %s):\n\t%s returned 0x%x (%s)", __LINE__, __FILE__, #fn, status, errstr);\
    exit(EXIT_FAILURE); \
  } \
} while (0)

#endif

// Check whether CUDA driver and CUDA device exist. Return 0 if exist, else return -1
int gpu_cuda_version_check()
{
    int driverVersion  = -1;

	hipError_t res = hipSuccess;

	res = hipDriverGetVersion ( &driverVersion );
    if(res != hipSuccess)
    {
        // checkLastCudaError();
        NVLOGE(TAG, "%s: hipDriverGetVersion failed", __func__);
        return -1;
    }
    else
    {
         NVLOGI(TAG, "%s: hipDriverGetVersion = %d ", __func__,driverVersion);    
    }

    // NVLOGC(TAG, "%s: driverVersion=%d runtimeVersion=%d", __func__, driverVersion, runtimeVersion);

    if(driverVersion > 0 )
    {
        return 0;
    }
    else
    {
        return -1;
    }
}



int nv_ipc_gpu_data_page_lock(void* phost, size_t size)
{
    if(gpu_cuda_version_check() < 0)
    {
        NVLOGE(TAG, "%s: CUDA driver or device not exist, skip", __func__);
        return -1;
    }

    unsigned int flag = hipHostRegisterPortable | hipHostRegisterMapped;
    hipError_t res = hipHostRegister (phost, size, flag);

    if( res != hipSuccess)
    {
        NVLOGE_NO(TAG, AERIAL_CUDA_API_EVENT, "%s: hipHostRegister failed!! res=%d", __func__,res);
        //CUDA_CHECK(res);
        return -1;
    }
    else
    {
        NVLOGI(TAG, "%s: OK", __func__);
        return 0;
    }
}


int nv_ipc_gpu_data_page_unlock(void* phost)
{
    if(gpu_cuda_version_check() < 0)
    {
        NVLOGI(TAG, "%s: CUDA driver or device not exist, skip", __func__);
        return -1;
    }

    if(hipHostUnregister (phost) != hipSuccess)
    {        
        NVLOGE_NO(TAG, AERIAL_CUDA_API_EVENT, "%s: hipHostUnregister failed", __func__);
        return -1;
    }
    else
    {
        NVLOGI(TAG, "%s: OK", __func__);
        return 0;
    }
}


int nv_ipc_gdrmemcpy_to_host(void* host, const void* device, size_t size)
{
#if 0
	int gdr_copy_from_mapping(gdr_mh_t handle, void *h_ptr, const void *map_d_ptr, size_t size)
#endif
    return 0;
}

int nv_ipc_gdrmemcpy_to_device(void* device, const void* host, size_t size)
{
#if 0
    gdr_copy_to_mapping(gdr_mh_t handle, void * map_d_ptr, const void * h_ptr, size_t size);
	gdr_copy_to_mapping_internal(void * map_d_ptr, const void * h_ptr, size_t size, int wc_mapping)
#endif
	return 0;
}
