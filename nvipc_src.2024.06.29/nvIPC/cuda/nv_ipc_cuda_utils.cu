/*
 * Copyright (c) 2020-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

#include <stdio.h>
#include <string.h>
#include <sys/mman.h>

#include "nv_ipc_cuda_utils.h"
#include "nv_ipc_utils.h"
// #include "nvlog.hpp"

#define TAG "NVIPC.CUDAUTILS"

inline hipError_t __checkLastCudaError(const char* file, int line)
{
    hipError_t lastErr = hipGetLastError();
    if(lastErr != hipSuccess)
    {
        NVLOGE_NO_FMT(TAG, AERIAL_CUDA_API_EVENT, "Error at {} line {}: {}", file, line, hipGetErrorString(lastErr));
    }
    return lastErr;
}
#define checkLastCudaError() __checkLastCudaError(__FILE__, __LINE__)

// Check whether CUDA driver and CUDA device exist. Return 0 if exist, else return -1
int cuda_version_check()
{
    int driverVersion  = -1;
    int runtimeVersion = -1;

    if(hipDriverGetVersion(&driverVersion) != hipSuccess)
    {
        // checkLastCudaError();
        // NVLOGI_FMT(TAG, "{}: hipDriverGetVersion failed", __func__);
    }
    else
    {
        if(hipRuntimeGetVersion(&runtimeVersion) != hipSuccess)
        {
            // checkLastCudaError();
            // NVLOGI_FMT(TAG, "{}: hipRuntimeGetVersion failed", __func__);
        }
    }

    // NVLOGC_FMT(TAG, "{}: driverVersion={} runtimeVersion={}", __func__, driverVersion, runtimeVersion);

    if(driverVersion > 0 && runtimeVersion > 0)
    {
        return 0;
    }
    else
    {
        return -1;
    }
}

int cuda_is_device_pointer(const void *ptr)
{
    int in_gpu = 0;

    hipPointerAttribute_t attr;
    attr.type = cudaMemoryTypeUnregistered;

    if(hipPointerGetAttributes(&attr, ptr) != hipSuccess)
    {
        NVLOGD_FMT(TAG, "{}: hipPointerGetAttributes failed", __func__);
        in_gpu = 0;
    }

    if(attr.type == hipMemoryTypeDevice) // Or hipMemoryTypeManaged?
    {
        in_gpu = 1;
    }

    NVLOGD_FMT(TAG, "{}: {} attr: type={} device={} devicePointer=0x{} hostPointer=0x{} in_gpu={}",
            __func__, (void *)ptr, attr.type, attr.device, attr.devicePointer, attr.hostPointer, in_gpu);
    return in_gpu;
}

int cuda_get_device_count(void)
{
    int num;
    hipError_t err = hipGetDeviceCount (&num);
    if (err != hipSuccess)
    {
        NVLOGW_FMT(TAG, "{}: hipGetDeviceCount failed", __func__);
        return -1;
    }
    else
    {
        return num;
    }
}

int cuda_page_lock(void* phost, size_t size)
{
    if(cuda_version_check() < 0)
    {
        NVLOGI_FMT(TAG, "{}: CUDA driver or device not exist, skip", __func__);
        return 0;
    }

    int flag = hipHostRegisterPortable | hipHostRegisterMapped;
    if(hipHostRegister(phost, size, flag) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_NO_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipHostRegister failed", __func__);
        return -1;
    }
    else
    {
        NVLOGI_FMT(TAG, "{}: OK", __func__);
        return 0;
    }
}

int cuda_page_unlock(void* phost)
{
    if(cuda_version_check() < 0)
    {
        NVLOGI_FMT(TAG, "{}: CUDA driver or device not exist, skip", __func__);
        return 0;
    }

    if(hipHostUnregister(phost) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_NO_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipHostUnregister failed", __func__);
        return -1;
    }
    else
    {
        NVLOGI_FMT(TAG, "{}: OK", __func__);
        return 0;
    }
}

int nv_ipc_memcpy_to_host(void* host, const void* device, size_t size)
{
    NVLOGV_FMT(TAG, "{}: dst_host={} src_gpu={} size={}", __func__, host, (void *)device, size);
/*
    if(hipSetDevice(0) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipSetDevice to {} failed", __func__, 0);
        return -1;
    }
*/
    if(hipMemcpy(host, device, size, hipMemcpyDeviceToHost) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_NO_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipMemcpy failed", __func__);
        return -1;
    }
    else
    {
        return 0;
    }
}

int nv_ipc_memcpy_to_device(void* device, const void* host, size_t size)
{
    NVLOGV_FMT(TAG, "{}: dst_gpu={} src_host={} size={}", __func__, device, (void *)host, size);
/*
    if(hipSetDevice(0) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipSetDevice to {} failed", __func__, 0);
        return -1;
    }
*/
    if(hipMemcpy(device, host, size, hipMemcpyHostToDevice) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_NO_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipMemcpy failed", __func__);
        return -1;
    }
    else
    {
        return 0;
    }
}
