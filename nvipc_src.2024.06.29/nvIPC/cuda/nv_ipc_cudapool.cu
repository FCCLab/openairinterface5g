/*
 * Copyright (c) 2020-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

#include <stdio.h>
#include <string.h>

#include "nv_ipc_cudapool.h"
#include "nv_ipc_utils.h"

#define TAG "NVIPC.CUDAPOOL"

#define CONFIG_CREATE_CUDA_STREAM 1

inline hipError_t __checkLastCudaError(const char* file, int line)
{
    hipError_t lastErr = hipGetLastError();
    if(lastErr != hipSuccess)
    {
        NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "Error at {} line {}: {}", file, line, hipGetErrorString(lastErr));
    }
    return lastErr;
}
#define checkLastCudaError() __checkLastCudaError(__FILE__, __LINE__)

typedef struct
{
    hipIpcMemHandle_t   memHandle;
    hipIpcEventHandle_t eventHandle;
} cuda_ipc_info_t;

typedef struct
{
    int primary;

    // CUDA device ID for CUDA memory case
    int device_id;

    size_t size;

    hipEvent_t event;

    hipStream_t stream;

    // For store the CUDA info communicated between different processes, should be in CPU SHM
    cuda_ipc_info_t* ipc_info;
    void*            cuda_addr;

} priv_data_t;

static inline priv_data_t* get_private_data(nv_ipc_cudapool_t* cudapool)
{
    return (priv_data_t*)((char*)cudapool + sizeof(nv_ipc_cudapool_t));
}

static int cudapool_create(priv_data_t* priv_data)
{
    if(hipMalloc(&priv_data->cuda_addr, priv_data->size) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: failed to allocate device memory", __func__);
        return -1;
    }

    memset(priv_data->ipc_info, 0, sizeof(cuda_ipc_info_t));

    if(hipIpcGetMemHandle(&priv_data->ipc_info->memHandle, priv_data->cuda_addr) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: failed to create memory handler", __func__);
        return -1;
    }

    if(hipEventCreateWithFlags(&priv_data->event, hipEventDisableTiming | hipEventInterprocess) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: failed to create event", __func__);
        return -1;
    }

    if(hipIpcGetEventHandle(&priv_data->ipc_info->eventHandle, priv_data->event) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: failed to get event handler", __func__);
        return -1;
    }
    else
    {
        NVLOGI_FMT(TAG, "{}: sizeof(cuda_ipc_info_t)={} device_id={} cuda_addr={} OK", __func__, sizeof(cuda_ipc_info_t), priv_data->device_id, priv_data->cuda_addr);
        return 0;
    }
}

static int cudapool_lookup(priv_data_t* priv_data)
{
    if(hipIpcOpenMemHandle(&priv_data->cuda_addr, priv_data->ipc_info->memHandle, hipIpcMemLazyEnablePeerAccess) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: failed to lookup memory handler", __func__);
        return -1;
    }

    if(hipIpcOpenEventHandle(&priv_data->event, priv_data->ipc_info->eventHandle) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: failed to lookup event handler", __func__);
        return -1;
    }
    else
    {
        //NVLOGI_FMT(TAG, "{}: sizeof(cuda_ipc_info_t)={} device_id={} cuda_addr={} OK", __func__, sizeof(cuda_ipc_info_t), priv_data->device_id, (void *)priv_data->cuda_addr);
        return 0;
    }
}

static int cudapool_close(priv_data_t* priv_data)
{
    if(hipIpcCloseMemHandle(priv_data->cuda_addr) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: failed to close memory handler", __func__);
        return -1;
    }
    else
    {
        NVLOGI_FMT(TAG, "{}: OK", __func__);
        return 0;
    }
}

static int cudapool_destroy(priv_data_t* priv_data)
{
    if(hipFree(priv_data->cuda_addr) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: failed to free memory", __func__);
        return -1;
    }
    else
    {
        NVLOGI_FMT(TAG, "{}: OK", __func__);
        return 0;
    }
}

static void* ipc_get_cudapool_addr(nv_ipc_cudapool_t* cudapool)
{
    if(cudapool == NULL)
    {
        NVLOGE_FMT(TAG, AERIAL_NVIPC_API_EVENT, "{}: instance not exist", __func__);
        return NULL;
    }
    priv_data_t* priv_data = get_private_data(cudapool);
    return priv_data->cuda_addr;
}

static int ipc_memcpy_to_host(nv_ipc_cudapool_t* cudapool, void* host, const void* device, size_t size)
{
    NVLOGV_FMT(TAG, "{}: dst_host={} src_gpu={} size={}", __func__, host, (void *)device, size);

    if(cudapool == NULL)
    {
        NVLOGE_FMT(TAG, AERIAL_NVIPC_API_EVENT, "{}: instance not exist", __func__);
        return -1;
    }

    priv_data_t* priv_data = get_private_data(cudapool);

    if(hipSetDevice(priv_data->device_id) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipSetDevice to {} failed", __func__, priv_data->device_id);
        return -1;
    }

    int ret = 0;

    if(CONFIG_CREATE_CUDA_STREAM)
    {
        if(hipMemcpyAsync(host, device, size, hipMemcpyDeviceToHost, priv_data->stream) != hipSuccess)
        {
            checkLastCudaError();
            NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipMemcpyAsync failed", __func__);
            ret = -1;
        }
        else if(hipStreamSynchronize(priv_data->stream) != hipSuccess)
        {
            checkLastCudaError();
            NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipStreamSynchronize failed", __func__);
            ret = -1;
        }
    }
    else
    {
        if(hipMemcpy(host, device, size, hipMemcpyDeviceToHost) != hipSuccess)
        {
            checkLastCudaError();
            NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipMemcpy failed", __func__);
            ret = -1;
        }
    }

    return ret;
}

static int ipc_memcpy_to_device(nv_ipc_cudapool_t* cudapool, void* device, const void* host, size_t size)
{
    NVLOGV_FMT(TAG, "{}: dst_gpu={} src_host={} size={}", __func__, device, (void *)host, size);

    if(cudapool == NULL)
    {
        NVLOGE_FMT(TAG, AERIAL_NVIPC_API_EVENT, "{}: instance not exist", __func__);
        return -1;
    }

    priv_data_t* priv_data = get_private_data(cudapool);

    if(hipSetDevice(priv_data->device_id) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipSetDevice to {} failed", __func__, priv_data->device_id);
        return -1;
    }

    int ret = 0;

    if(CONFIG_CREATE_CUDA_STREAM)
    {
        if(hipMemcpyAsync(device, host, size, hipMemcpyHostToDevice, priv_data->stream) != hipSuccess)
        {
            checkLastCudaError();
            NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipMemcpyAsync failed", __func__);
            ret = -1;
        }
        else if(hipStreamSynchronize(priv_data->stream) != hipSuccess)
        {
            checkLastCudaError();
            NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipStreamSynchronize failed", __func__);
            ret = -1;
        }
    }
    else
    {
        if(hipMemcpy(device, host, size, hipMemcpyHostToDevice) != hipSuccess)
        {
            checkLastCudaError();
            NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipMemcpy failed", __func__);
            ret = -1;
        }
    }

    return ret;
}

static int ipc_cudapool_close(nv_ipc_cudapool_t* cudapool)
{
    if(cudapool == NULL)
    {
        NVLOGE_FMT(TAG, AERIAL_NVIPC_API_EVENT, "{}: instance not exist", __func__);
        return -1;
    }

    priv_data_t* priv_data = get_private_data(cudapool);

    int ret = 0;
    if(hipSetDevice(priv_data->device_id) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipSetDevice to {} failed", __func__, priv_data->device_id);
        ret = -1;
    }

    if(CONFIG_CREATE_CUDA_STREAM)
    {
        if(hipStreamDestroy(priv_data->stream) != hipSuccess)
        {
            NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipStreamDestroy failed", __func__);
            ret = -1;
        }
    }

    if(priv_data->primary)
    {
        ret = cudapool_destroy(priv_data);
    }
    else
    {
        ret = cudapool_close(priv_data);
    }

    free(cudapool);

    if(ret == 0)
    {
        NVLOGI_FMT(TAG, "{}: OK", __func__);
    }
    return ret;
}

static int ipc_cudapool_open(priv_data_t* priv_data)
{
    if(hipSetDevice(priv_data->device_id) != hipSuccess)
    {
        checkLastCudaError();
        NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipSetDevice to {} failed", __func__, priv_data->device_id);
        return -1;
    }

    if(CONFIG_CREATE_CUDA_STREAM)
    {
        if(hipStreamCreate(&priv_data->stream) != hipSuccess)
        {
            NVLOGE_FMT(TAG, AERIAL_CUDA_API_EVENT, "{}: hipStreamCreate failed", __func__);
            return -1;
        }
    }

    if(priv_data->primary)
    {
        return cudapool_create(priv_data);
    }
    else
    {
        return cudapool_lookup(priv_data);
    }
}

nv_ipc_cudapool_t* nv_ipc_cudapool_open(int primary, void* shm, size_t size, int device_id)
{
    if(shm == NULL || size <= 0 || device_id < 0)
    {
        NVLOGE_FMT(TAG, AERIAL_NVIPC_API_EVENT, "{}: invalid parameter", __func__);
        return NULL;
    }

    int                struct_size = sizeof(nv_ipc_cudapool_t) + sizeof(priv_data_t);
    nv_ipc_cudapool_t* cudapool    = (nv_ipc_cudapool_t*)malloc(struct_size);
    if(cudapool == NULL)
    {
        NVLOGE_FMT(TAG, AERIAL_SYSTEM_API_EVENT, "{}: memory malloc failed", __func__);
        return NULL;
    }
    memset(cudapool, 0, struct_size);

    priv_data_t* priv_data = get_private_data(cudapool);
    priv_data->primary     = primary;
    priv_data->device_id   = device_id;
    priv_data->size        = size;
    priv_data->ipc_info    = (cuda_ipc_info_t*)shm;

    cudapool->get_cudapool_addr = ipc_get_cudapool_addr;
    cudapool->memcpy_to_host    = ipc_memcpy_to_host;
    cudapool->memcpy_to_device  = ipc_memcpy_to_device;
    cudapool->close             = ipc_cudapool_close;

    if(ipc_cudapool_open(priv_data) < 0)
    {
        NVLOGE_FMT(TAG, AERIAL_NVIPC_API_EVENT, "{}: Failed", __func__);
        ipc_cudapool_close(cudapool);
        return NULL;
    }
    else
    {
        NVLOGI_FMT(TAG, "{}: OK", __func__);
        return cudapool;
    }
}
